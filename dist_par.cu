// Aluno: Matheus Vieira de Souza       RGA: 2021.1904.048-3
// Disciplina: Programação Paralela
// Distância de edição paralela
// Para compilar: nvcc dist_par.cu -o dist_par
// Para executar: ./dist_par <nome_arquivo_entrada.txt>
// OBS: Somente a solução inicial (com grid de um único bloco) está completa, 
// tentei fazer a solução completa mas contém erros.


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

char *aloca_sequencia(int n)
{
	char *seq;

	seq = (char *)malloc((n + 1) * sizeof(char));
	if (seq == NULL)
	{
		printf("\nErro na alocação de estruturas\n");
		exit(1);
	}
	return seq;
}

int *aloca_matriz(int n)
{
	int *seq;

	seq = (int *)malloc((n) * sizeof(int));
	if (seq == NULL)
	{
		printf("\nErro na alocação de estruturas\n");
		exit(1);
	}
	return seq;
}

void libera(int n, char *s, char *r, int *d)
{
	free(s);
	free(r);
	free(d);
}

//----------------------------------------------
__global__ void distancia(int *d, int n, int m, int i, char *s, char *r)
{

	int posi, t, a, b, c, min;
	int cima, diag, atras;
	int linha, coluna;
	__syncthreads();

	if (i >= n)
	{
		posi = (i * (m + 1)) - ((i - n) * (m));
		posi = posi - threadIdx.x * (m);
	}
	else
		posi = i * (m + 1) - threadIdx.x * (m) + m + 2;

	atras = posi - 1;
	cima = posi - (m + 1);
	diag = posi - (m + 2);

	// Se é uma célula válida
	if ((d[posi] == 0 && posi > 0) && posi <= (n + 1) * (m + 1))
	{
		linha = (posi / (m + 1));
		coluna = posi - ((m + 1) * linha);

		t = (s[linha] != r[coluna] ? 1 : 0);
		a = d[atras] + 1;
		b = d[cima] + 1;
		c = d[diag] + t;

		// Calcula d[(i*(m+1)) + j] = min(a, b, c)
		if (a < b)
			min = a;
		else
			min = b;
		if (c < min)
			min = c;

		d[posi] = min;
	}
}

int main(int argc, char **argv)
{
	int n,	// Tamanho da sequência s
		m,	// Tamanho da sequência r
		*d, // Matriz de distâncias com tamanho (n+1)*(m+1)
		i, j;
	char *s,		  // Sequência s de entrada (vetor com tamanho n+1)
		*r;			  // Sequência r de entrada (vetor com tamanho m+1)
	FILE *arqEntrada; // Arquivo texto de entrada

	if (argc != 2)
	{
		printf("O programa foi executado com argumentos incorretos.\n");
		printf("Uso: ./dist_seq <nome arquivo entrada>\n");
		exit(1);
	}

	// Abre arquivo de entrada
	arqEntrada = fopen(argv[1], "rt");

	if (arqEntrada == NULL)
	{
		printf("\nArquivo texto de entrada não encontrado\n");
		exit(1);
	}

	// Lê tamanho das sequências s e r
	fscanf(arqEntrada, "%d %d", &n, &m);

	// Aloca vetores s e r
	s = aloca_sequencia(n);
	r = aloca_sequencia(m);
	// Aloca matriz d
	d = aloca_matriz((n + 1) * (m + 1));

	// Lê sequências do arquivo de entrada
	s[0] = ' ';
	r[0] = ' ';
	fscanf(arqEntrada, "%s", &(s[1]));
	fscanf(arqEntrada, "%s", &(r[1]));

	// Fecha arquivo de entrada
	fclose(arqEntrada);

	struct timeval h_ini; 
    struct timeval h_fim;
	gettimeofday(&h_ini, 0);

	// Inicializa matriz de distâncias d
	for (i = 0; i <= m; i++)
		d[i] = i;

	for (j = 1; j <= n; j++)
		d[(m * j) + j] = j;

	// Calcula distância de edição entre sequências s e r, por anti-diagonais
	int *d_M;
	char *d_s, *d_r;

	hipMalloc((void **)&d_M, sizeof(int) * ((n + 1) * (m + 1)));
	hipMalloc((void **)&d_s, sizeof(char) * (n + 1));
	hipMalloc((void **)&d_r, sizeof(char) * (m + 1));

	hipMemcpy(d_M, d, sizeof(int) * ((n + 1) * (m + 1)), hipMemcpyHostToDevice);
	hipMemcpy(d_s, s, sizeof(char) * (n + 1), hipMemcpyHostToDevice);
	hipMemcpy(d_r, r, sizeof(char) * (m + 1), hipMemcpyHostToDevice);

	for (int i = 0; i < n + m + 1; i++)
		distancia<<<1, n>>>(d_M, n, m, i, d_s, d_r);

	hipDeviceSynchronize();

	hipMemcpy(d, d_M, sizeof(int) * ((n + 1) * (m + 1)), hipMemcpyDeviceToHost);

	gettimeofday(&h_fim, 0);
	
    // Tempo de execução na CPU em milissegundos
	long segundos = h_fim.tv_sec - h_ini.tv_sec;
	long milisegundos = h_fim.tv_usec - h_ini.tv_usec;
	double tempo = (segundos * 1e3) + (milisegundos * 1e-3);

	printf("Distância = %d\n", d[((n + 1) * (m + 1)) - 1]);
	printf("Tempo CPU = %.2f ms\n", tempo);

	// Libera vetores s e r e matriz d
	libera(n, s, r, d);

	return 0;
}